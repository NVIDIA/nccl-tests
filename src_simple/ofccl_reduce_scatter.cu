#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "common_simple.h"
#include <stdio.h>
#include <unistd.h>
#include <pthread.h>
#include <sched.h>

void print_header() {
  PRINT("# %10s  %12s  %8s  %6s            out-of-place                       in-place          \n", "", "", "", "");
  PRINT("# %10s  %12s  %8s  %6s  %7s  %6s  %6s  %5s  %7s  %6s  %6s  %5s\n", "size", "count", "type", "redop",
        "time", "algbw", "busbw", "error", "time", "algbw", "busbw", "error");
  PRINT("# %10s  %12s  %8s  %6s  %7s  %6s  %6s  %5s  %7s  %6s  %6s  %5s\n", "(B)", "(elements)", "", "",
        "(us)", "(GB/s)", "(GB/s)", "", "(us)", "(GB/s)", "(GB/s)", "");
}

void print_line_header (size_t size, size_t count, const char *typeName, const char *opName, int root) {
  PRINT("%12li  %12li  %8s  %6s", size, count, typeName, opName);
}

void ReduceScatterGetCollByteCount(size_t *sendcount, size_t *recvcount, size_t *paramcount, size_t *sendInplaceOffset, size_t *recvInplaceOffset, size_t count, int nranks) {
  *sendcount = (count/nranks)*nranks;
  *recvcount = count/nranks;
  *sendInplaceOffset = 0;
  *recvInplaceOffset = count/nranks;
  *paramcount = *recvcount;
}

testResult_t ReduceScatterInitData(struct threadArgs* args, ncclDataType_t type, ncclRedOp_t op, int root, int rep, int in_place) {
  size_t sendcount = args->sendBytes / wordSize(type);
  size_t recvcount = args->expectedBytes / wordSize(type);
  int nranks = args->nProcs*args->nThreads*args->nGpus;

  for (int i=0; i<args->nGpus; i++) {
    int gpuid = args->localRank*args->nThreads*args->nGpus + args->thread*args->nGpus + i;
    CUDACHECK(hipSetDevice(gpuid));
    int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
    CUDACHECK(hipMemset(args->recvbuffs[i], 0, args->expectedBytes));
    void* data = in_place ? args->recvbuffs[i] : args->sendbuffs[i];
    TESTCHECK(InitData(data, sendcount, type, rep, rank));
    CUDACHECK(hipMemcpy(args->expected[i], args->recvbuffs[i], args->expectedBytes, hipMemcpyDefault));
    TESTCHECK(InitDataReduce(args->expected[i], recvcount, rank*recvcount, type, op, rep, nranks));
    CUDACHECK(hipDeviceSynchronize());
  }
  return testSuccess;
}

void ReduceScatterGetBw(size_t count, int typesize, double sec, double* algBw, double* busBw, int nranks) {
  double baseBw = (double)(count * typesize * nranks) / 1.0E9 / sec;

  *algBw = baseBw;
  double factor = ((double)(nranks - 1))/((double)nranks);
  *busBw = baseBw * factor;
}

int myCallback(int collIdFromCqe, void *args) {
  // 不打log把这里删了，不然影响性能。
  // if (collId != collIdFromCqe) {
  //   // more robust error handle.
  //   OFTEST_LOG(TEST_ERROR, "<%lu> Rank<%d>, collIdFromCqe(%d) is not expected(%d)", pthread_self(), cudaDev, collIdFromCqe, collId);
  //   return -1;
  // }
  pthread_mutex_lock(&(((CallBackArgs *)args)->mutex));
  ((CallBackArgs *)args)->gotCqe = 1;

  // int cudaDev;
  // CUDACHECK(hipGetDevice(&cudaDev));
  // int collId = ((CallBackArgs *)args)->collId;
  // OFTEST_LOG(TEST, "<%lu> Rank<%d>, callback get cqe for coll_id = %d", pthread_self(), cudaDev, collId);
  // OFTEST_LOG(TEST, "<%lu> Rank<%d>, callback get %dth cqe for coll_id = %d", pthread_self(), cudaDev, ((CallBackArgs *)args)->cqeCnt++, collId);

  pthread_mutex_unlock(&(((CallBackArgs *)args)->mutex));
  return 0;
}

testResult_t ReduceScatterRunColl(void* sendbuff, void* recvbuff, int collId, CallBackArgs *args, ofcclRankCtx_t rankCtx) {
  args->collId = collId;
  args->gotCqe = 0;
  pthread_mutex_init(&args->mutex, NULL);
  NCCLCHECK(ofcclRunReduceScatter(sendbuff, recvbuff, collId, myCallback, args, rankCtx));

  // int cudaDev;
  // CUDACHECK(hipGetDevice(&cudaDev));
  // OFTEST_LOG(TEST, "<%lu> Rank<%d>, invoke ofcclRunReduceScatter for coll_id = %d with args @ %p", pthread_self(), cudaDev, collId, args);
  // OFTEST_LOG(TEST, "<%lu> Rank<%d>, invoke ofcclRunReduceScatter sendbuff @ %p, recvbuff @ %p", pthread_self(), cudaDev, sendbuff, recvbuff);
  
  return testSuccess;
}

testResult_t ReduceScatterPrepare(size_t count, ncclDataType_t datatype, ncclRedOp_t op, ncclComm* comm, int collId, ofcclRankCtx_t rankCtx) {

  NCCLCHECK(ofcclPrepareReduceScatter(count, datatype, op, comm, collId, rankCtx));
  // OFTEST_LOG(TEST, "tid<%lu> invoke ofcclPrepareReduceScatter with count=%lu, collId=%d", pthread_self(), count, collId);
  return testSuccess;
}

struct testColl reduceScatterTest = {
  "ReduceScatter",
  ReduceScatterGetCollByteCount,
  ReduceScatterInitData,
  ReduceScatterGetBw,
  ReduceScatterRunColl,
  ReduceScatterPrepare
};

void ReduceScatterGetBuffSize(size_t *sendcount, size_t *recvcount, size_t count, int nranks) {
  size_t paramcount, sendInplaceOffset, recvInplaceOffset;
  ReduceScatterGetCollByteCount(sendcount, recvcount, &paramcount, &sendInplaceOffset, &recvInplaceOffset, count, nranks);
}

testResult_t ReduceScatterRunTest(struct threadArgs* args, int root, ncclDataType_t type, const char* typeName, ncclRedOp_t op, const char* opName) {
  args->collTest = &reduceScatterTest;
  ncclDataType_t *run_types;
  ncclRedOp_t *run_ops;
  const char **run_typenames, **run_opnames;
  int type_count, op_count;

  if ((int)type != -1) {
    type_count = 1;
    run_types = &type;
    run_typenames = &typeName;
  } else {
    type_count = test_typenum;
    run_types = test_types;
    run_typenames = test_typenames;
  }

  if ((int)op != -1) {
    op_count = 1;
    run_ops = &op;
    run_opnames = &opName;
  } else {
    op_count = test_opnum;
    run_ops = test_ops;
    run_opnames = test_opnames;
  }

  for (int i=0; i<type_count; i++) {
    for (int j=0; j<op_count; j++) {
      TESTCHECK(TimeTest(args, run_types[i], run_typenames[i], run_ops[j], run_opnames[j], -1, true));
    }
  }
  return testSuccess;
}

struct testEngine reduceScatterEngine = {
  ReduceScatterGetBuffSize,
  ReduceScatterRunTest
};

#pragma weak ncclTestEngine=reduceScatterEngine



