/*************************************************************************
 * Copyright (c) 2016-2022, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "hip/hip_runtime.h"
#include "common.h"

void ReduceGetCollByteCount(size_t *sendcount, size_t *recvcount, size_t *paramcount, size_t *sendInplaceOffset, size_t *recvInplaceOffset, size_t count, int nranks) {
  *sendcount = count;
  *recvcount = count;
  *sendInplaceOffset = 0;
  *recvInplaceOffset = 0;
  *paramcount = *sendcount;
}

testResult_t ReduceInitData(struct threadArgs* args, ncclDataType_t type, ncclRedOp_t op, int root, int rep, int in_place) {
  size_t sendcount = args->sendBytes / wordSize(type);
  size_t recvcount = args->expectedBytes / wordSize(type);
  int nranks = args->nProcs*args->nThreads*args->nGpus;

  for (int i=0; i<args->nGpus; i++) {
    CUDACHECK(hipSetDevice(args->gpus[i]));
    int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
    CUDACHECK(hipMemset(args->recvbuffs[i], 0, args->expectedBytes));
    void* data = in_place ? args->recvbuffs[i] : args->sendbuffs[i];
    TESTCHECK(InitData(data, sendcount, 0, type, op, rep, nranks, rank));
    CUDACHECK(hipMemcpy(args->expected[i], args->recvbuffs[i], args->expectedBytes, hipMemcpyDefault));
    if (rank == root) TESTCHECK(InitDataReduce(args->expected[i], recvcount, 0, type, op, rep, nranks));
    CUDACHECK(hipDeviceSynchronize());
  }
  return testSuccess;
}

void ReduceGetBw(size_t count, int typesize, double sec, double* algBw, double* busBw, int nranks) {
  double baseBw = (double)(count * typesize) / 1.0E9 / sec;
  *algBw = baseBw;
  *busBw = baseBw;
}

testResult_t ReduceRunColl(void* sendbuff, void* recvbuff, size_t count, ncclDataType_t type, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream, struct threadArgs* args) {
  NCCLCHECK(ncclReduce(sendbuff, recvbuff, count, type, op, root, comm, stream));
  return testSuccess;
}

struct testColl reduceTest = {
  "Reduce",
  ReduceGetCollByteCount,
  ReduceInitData,
  ReduceGetBw,
  ReduceRunColl
};

void ReduceGetBuffSize(size_t *sendcount, size_t *recvcount, size_t count, int nranks) {
  size_t paramcount, sendInplaceOffset, recvInplaceOffset;
  ReduceGetCollByteCount(sendcount, recvcount, &paramcount, &sendInplaceOffset, &recvInplaceOffset, count, nranks);
}

testResult_t ReduceRunTest(struct threadArgs* args, int root, ncclDataType_t type, const char* typeName, ncclRedOp_t op, const char* opName) {
  args->collTest = &reduceTest;
  ncclDataType_t *run_types;
  ncclRedOp_t *run_ops;
  const char **run_typenames, **run_opnames;
  int type_count, op_count;
  int begin_root, end_root;

  if ((int)type != -1) {
    type_count = 1;
    run_types = &type;
    run_typenames = &typeName;
  } else {
    type_count = test_typenum;
    run_types = test_types;
    run_typenames = test_typenames;
  }

  if ((int)op != -1) {
    op_count = 1;
    run_ops = &op;
    run_opnames = &opName;
  } else {
    op_count = test_opnum;
    run_ops = test_ops;
    run_opnames = test_opnames;
  }

  if (root != -1) {
    begin_root = end_root = root;
  } else {
    begin_root = 0;
    end_root = args->nProcs*args->nThreads*args->nGpus-1;
  }

  for (int i=0; i<type_count; i++) {
    for (int j=0; j<op_count; j++) {
      for (int k=begin_root; k<=end_root; k++) {
        TESTCHECK(TimeTest(args, run_types[i], run_typenames[i], run_ops[j], run_opnames[j], k));
      }
    }
  }
  return testSuccess;
}

struct testEngine reduceEngine = {
  ReduceGetBuffSize,
  ReduceRunTest
};

#pragma weak ncclTestEngine=reduceEngine
