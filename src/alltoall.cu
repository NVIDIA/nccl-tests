/*************************************************************************
 * Copyright (c) 2016-2020, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "hip/hip_runtime.h"
#include "common.h"

void print_header() {
  PRINT("# %10s  %12s  %6s  %6s            out-of-place                       in-place          \n", "", "", "", "");
  PRINT("# %10s  %12s  %6s  %6s  %7s  %6s  %6s  %5s  %7s  %6s  %6s  %5s\n", "size", "count", "type", "redop",
        "time", "algbw", "busbw", "error", "time", "algbw", "busbw", "error");
  PRINT("# %10s  %12s  %6s  %6s  %7s  %6s  %6s  %5s  %7s  %6s  %6s  %5s\n", "(B)", "(elements)", "", "",
        "(us)", "(GB/s)", "(GB/s)", "", "(us)", "(GB/s)", "(GB/s)", "");
}

void print_line_header (size_t size, size_t count, const char *typeName, const char *opName, int root) {
  PRINT("%12li  %12li  %6s  %6s", size, count, typeName, opName);
}

void AlltoAllGetCollByteCount(size_t *sendcount, size_t *recvcount, size_t *paramcount, size_t *sendInplaceOffset, size_t *recvInplaceOffset, size_t count, int nranks) {
  *sendcount = (count/nranks)*nranks;
  *recvcount = (count/nranks)*nranks;
  *sendInplaceOffset = 0;
  *recvInplaceOffset = 0;
  *paramcount = count/nranks;
}

testResult_t AlltoAllInitData(struct threadArgs* args, ncclDataType_t type, ncclRedOp_t op, int root, int rep, int in_place) {
  size_t sendcount = args->sendBytes / wordSize(type);
  size_t recvcount = args->expectedBytes / wordSize(type);
  int nranks = args->nProcs*args->nThreads*args->nGpus;

  for (int i=0; i<args->nGpus; i++) {
    char* str = getenv("NCCL_TESTS_DEVICE");
    int gpuid = str ? atoi(str) : args->localRank*args->nThreads*args->nGpus + args->thread*args->nGpus + i;
    CUDACHECK(hipSetDevice(gpuid));
    int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
    CUDACHECK(hipMemset(args->recvbuffs[i], 0, args->expectedBytes));
    void* data = in_place ? args->recvbuffs[i] : args->sendbuffs[i];
    TESTCHECK(InitData(data, sendcount, type, rep, rank));
    for (int j=0; j<nranks; j++) {
      TESTCHECK(InitData(((char*)args->expected[i])+args->sendBytes/nranks*j, sendcount/nranks, type, rep+rank*sendcount/nranks, j));
    }
    CUDACHECK(hipDeviceSynchronize());
  }
  // We don't support in-place alltoall
  args->reportErrors = in_place ? 0 : 1;
  return testSuccess;
}

void AlltoAllGetBw(size_t count, int typesize, double sec, double* algBw, double* busBw, int nranks) {
  double baseBw = (double)(count * nranks * typesize) / 1.0E9 / sec;

  *algBw = baseBw;
  double factor = ((double)(nranks-1))/((double)(nranks));
  *busBw = baseBw * factor;
}

testResult_t AlltoAllRunColl(void* sendbuff, void* recvbuff, size_t count, ncclDataType_t type, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream) {
  int nRanks;
  NCCLCHECK(ncclCommCount(comm, &nRanks));
  size_t rankOffset = count * wordSize(type);
  if (count == 0) return testSuccess;

#if NCCL_MAJOR < 2 || NCCL_MINOR < 7
  printf("NCCL 2.7 or later is needed for alltoall. This test was compiled with %d.%d.\n", NCCL_MAJOR, NCCL_MINOR);
  return testNcclError;
#else
  NCCLCHECK(ncclAllToAll(sendbuff, recvbuff, count, type, comm, stream));
  // NCCLCHECK(ncclGroupStart());
  // for (int r=0; r<nRanks; r++) {
  //   NCCLCHECK(ncclSend(((char*)sendbuff)+r*rankOffset, count, type, r, comm, stream));
  //   NCCLCHECK(ncclRecv(((char*)recvbuff)+r*rankOffset, count, type, r, comm, stream));
  // }
  // NCCLCHECK(ncclGroupEnd());
  return testSuccess;
#endif
}

struct testColl alltoAllTest = {
  "AlltoAll",
  AlltoAllGetCollByteCount,
  AlltoAllInitData,
  AlltoAllGetBw,
  AlltoAllRunColl
};

void AlltoAllGetBuffSize(size_t *sendcount, size_t *recvcount, size_t count, int nranks) {
  size_t paramcount, sendInplaceOffset, recvInplaceOffset;
  AlltoAllGetCollByteCount(sendcount, recvcount, &paramcount, &sendInplaceOffset, &recvInplaceOffset, count, nranks);
}

testResult_t AlltoAllRunTest(struct threadArgs* args, int root, ncclDataType_t type, const char* typeName, ncclRedOp_t op, const char* opName) {
  args->collTest = &alltoAllTest;
  ncclDataType_t *run_types;
  const char **run_typenames;
  int type_count;

  if ((int)type != -1) {
    type_count = 1;
    run_types = &type;
    run_typenames = &typeName;
  } else {
    type_count = ncclNumTypes;
    run_types = test_types;
    run_typenames = test_typenames;
  }

  for (int i=0; i<type_count; i++) {
      TESTCHECK(TimeTest(args, run_types[i], run_typenames[i], (ncclRedOp_t)0, "", -1));
  }
  return testSuccess;
}

struct testEngine alltoAllEngine = {
  AlltoAllGetBuffSize,
  AlltoAllRunTest
};

#pragma weak ncclTestEngine=alltoAllEngine
