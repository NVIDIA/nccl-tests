/*************************************************************************
 * Copyright (c) 2016-2022, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "hip/hip_runtime.h"
#include "common.h"

void AlltoAllGetCollByteCount(size_t *sendcount, size_t *recvcount, size_t *paramcount, size_t *sendInplaceOffset, size_t *recvInplaceOffset, size_t count, int nranks) {
  *sendcount = (count/nranks)*nranks;
  *recvcount = (count/nranks)*nranks;
  *sendInplaceOffset = 0;
  *recvInplaceOffset = 0;
  *paramcount = count/nranks;
}

testResult_t AlltoAllInitData(struct threadArgs* args, ncclDataType_t type, ncclRedOp_t op, int root, int rep, int in_place) {
  size_t sendcount = args->sendBytes / wordSize(type);
  size_t recvcount = args->expectedBytes / wordSize(type);
  int nranks = args->nProcs*args->nThreads*args->nGpus;

  for (int i=0; i<args->nGpus; i++) {
    CUDACHECK(hipSetDevice(args->gpus[i]));
    int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
    CUDACHECK(hipMemset(args->recvbuffs[i], 0, args->expectedBytes));
    void* data = in_place ? args->recvbuffs[i] : args->sendbuffs[i];
    TESTCHECK(InitData(data, sendcount, 0, type, ncclSum, 33*rep + rank, 1, 0));
    for (int j=0; j<nranks; j++) {
      size_t partcount = sendcount/nranks;
      TESTCHECK(InitData((char*)args->expected[i] + j*partcount*wordSize(type), partcount, rank*partcount, type, ncclSum, 33*rep + j, 1, 0));
    }
    CUDACHECK(hipDeviceSynchronize());
  }
  // We don't support in-place alltoall
  args->reportErrors = in_place ? 0 : 1;
  return testSuccess;
}

void AlltoAllGetBw(size_t count, int typesize, double sec, double* algBw, double* busBw, int nranks) {
  double baseBw = (double)(count * nranks * typesize) / 1.0E9 / sec;

  *algBw = baseBw;
  double factor = ((double)(nranks-1))/((double)(nranks));
  *busBw = baseBw * factor;
}

testResult_t AlltoAllRunColl(void* sendbuff, void* recvbuff, size_t count, ncclDataType_t type, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream, struct threadArgs* args) {
  int nRanks;
  NCCLCHECK(ncclCommCount(comm, &nRanks));
  size_t rankOffset = count * wordSize(type);

#if NCCL_MAJOR < 2 || NCCL_MINOR < 7
  printf("NCCL 2.7 or later is needed for alltoall. This test was compiled with %d.%d.\n", NCCL_MAJOR, NCCL_MINOR);
  return testNcclError;
#else
  NCCLCHECK(ncclGroupStart());
  for (int r=0; r<nRanks; r++) {
    NCCLCHECK(ncclSend(((char*)sendbuff)+r*rankOffset, count, type, r, comm, stream));
    NCCLCHECK(ncclRecv(((char*)recvbuff)+r*rankOffset, count, type, r, comm, stream));
  }
  NCCLCHECK(ncclGroupEnd());
  return testSuccess;
#endif
}

struct testColl alltoAllTest = {
  "AlltoAll",
  AlltoAllGetCollByteCount,
  AlltoAllInitData,
  AlltoAllGetBw,
  AlltoAllRunColl
};

void AlltoAllGetBuffSize(size_t *sendcount, size_t *recvcount, size_t count, int nranks) {
  size_t paramcount, sendInplaceOffset, recvInplaceOffset;
  AlltoAllGetCollByteCount(sendcount, recvcount, &paramcount, &sendInplaceOffset, &recvInplaceOffset, count, nranks);
}

testResult_t AlltoAllRunTest(struct threadArgs* args, int root, ncclDataType_t type, const char* typeName, ncclRedOp_t op, const char* opName) {
  args->collTest = &alltoAllTest;
  ncclDataType_t *run_types;
  const char **run_typenames;
  int type_count;

  if ((int)type != -1) {
    type_count = 1;
    run_types = &type;
    run_typenames = &typeName;
  } else {
    type_count = test_typenum;
    run_types = test_types;
    run_typenames = test_typenames;
  }

  for (int i=0; i<type_count; i++) {
      TESTCHECK(TimeTest(args, run_types[i], run_typenames[i], (ncclRedOp_t)0, "none", -1));
  }
  return testSuccess;
}

struct testEngine alltoAllEngine = {
  AlltoAllGetBuffSize,
  AlltoAllRunTest
};

#pragma weak ncclTestEngine=alltoAllEngine
