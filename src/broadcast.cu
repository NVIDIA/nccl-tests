/*************************************************************************
 * Copyright (c) 2015-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "hip/hip_runtime.h"
#include "common.h"

void print_header() {
  PRINT("# %10s  %12s  %8s  %6s            out-of-place                       in-place          \n", "", "", "", "");
  PRINT("# %10s  %12s  %8s  %6s  %7s  %6s  %6s  %5s  %7s  %6s  %6s  %5s\n", "size", "count", "type", "root",
        "time", "algbw", "busbw", "error", "time", "algbw", "busbw", "error");
  PRINT("# %10s  %12s  %8s  %6s  %7s  %6s  %6s  %5s  %7s  %6s  %6s  %5s\n", "(B)", "(elements)", "", "",
        "(us)", "(GB/s)", "(GB/s)", "", "(us)", "(GB/s)", "(GB/s)", "");
}

void print_line_header (size_t size, size_t count, const char *typeName, const char *opName, int root) {
  PRINT("%12li  %12li  %8s  %6i", size, count, typeName, root);
}

void BroadcastGetCollByteCount(size_t *sendcount, size_t *recvcount, size_t *paramcount, size_t *sendInplaceOffset, size_t *recvInplaceOffset, size_t count, int nranks) {
  *sendcount = count;
  *recvcount = count;
  *sendInplaceOffset = 0;
  *recvInplaceOffset = 0;
  *paramcount = *sendcount;
}

testResult_t BroadcastInitData(struct threadArgs* args, ncclDataType_t type, ncclRedOp_t op, int root, int rep, int in_place) {
  size_t sendcount = args->sendBytes / wordSize(type);
  size_t recvcount = args->expectedBytes / wordSize(type);

  for (int i=0; i<args->nGpus; i++) {
    int gpuid = args->localRank*args->nThreads*args->nGpus + args->thread*args->nGpus + i;
    CUDACHECK(hipSetDevice(gpuid));
    int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
    CUDACHECK(hipMemset(args->recvbuffs[i], 0, args->expectedBytes));
    void* data = in_place ? args->recvbuffs[i] : args->sendbuffs[i];
    if (rank == root) TESTCHECK(InitData(data, sendcount, type, rep, rank));
    TESTCHECK(InitData(args->expected[i], recvcount, type, rep, root));
    CUDACHECK(hipDeviceSynchronize());
  }
  return testSuccess;
}

void BroadcastGetBw(size_t count, int typesize, double sec, double* algBw, double* busBw, int nranks) {
  double baseBw = (double)(count * typesize) / 1.0E9 / sec;

  *algBw = baseBw;
  double factor = 1;
  *busBw = baseBw * factor;
}

testResult_t BroadcastRunColl(void* sendbuff, void* recvbuff, size_t count, ncclDataType_t type, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream) {
  int rank;
  NCCLCHECK(ncclCommUserRank(comm, &rank));
#if NCCL_MAJOR >= 2 && NCCL_MINOR >= 2
  NCCLCHECK(ncclBroadcast(sendbuff, recvbuff, count, type, root, comm, stream));
#else
  if (rank == root) {
      NCCLCHECK(ncclBcast(sendbuff, count, type, root, comm, stream));
  } else {
      NCCLCHECK(ncclBcast(recvbuff, count, type, root, comm, stream));
  }
#endif
  return testSuccess;
}

struct testColl broadcastTest = {
  "Broadcast",
  BroadcastGetCollByteCount,
  BroadcastInitData,
  BroadcastGetBw,
  BroadcastRunColl
};

void BroadcastGetBuffSize(size_t *sendcount, size_t *recvcount, size_t count, int nranks) {
  size_t paramcount, sendInplaceOffset, recvInplaceOffset;
  BroadcastGetCollByteCount(sendcount, recvcount, &paramcount, &sendInplaceOffset, &recvInplaceOffset, count, nranks);
}

testResult_t BroadcastRunTest(struct threadArgs* args, int root, ncclDataType_t type, const char* typeName, ncclRedOp_t op, const char* opName) {
  args->collTest = &broadcastTest;
  ncclDataType_t *run_types;
  const char **run_typenames;
  int type_count;
  int begin_root, end_root;
  int step = 0;

  if ((int)type != -1) {
    type_count = 1;
    run_types = &type;
    run_typenames = &typeName;
  } else {
    type_count = test_typenum;
    run_types = test_types;
    run_typenames = test_typenames;
  }

  if (root >= 0) {
    begin_root = end_root = root;
  } else {
    step = -root;
    begin_root = 0;
    end_root = args->nProcs*args->nThreads*args->nGpus-1;
  }

  for (int i=0; i<type_count; i++) {
    for (int j=begin_root; j<=end_root; j+=step) {
      TESTCHECK(TimeTest(args, run_types[i], run_typenames[i], (ncclRedOp_t)0, "", j));
    }
  }
  return testSuccess;
}

struct testEngine broadcastEngine = {
  BroadcastGetBuffSize,
  BroadcastRunTest
};

#pragma weak ncclTestEngine=broadcastEngine
