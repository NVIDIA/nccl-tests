/*************************************************************************
 * Copyright (c) 2015-2022, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "hip/hip_runtime.h"
#include "common.h"

void BroadcastGetCollByteCount(size_t *sendcount, size_t *recvcount, size_t *paramcount, size_t *sendInplaceOffset, size_t *recvInplaceOffset, size_t count, int nranks) {
  *sendcount = count;
  *recvcount = count;
  *sendInplaceOffset = 0;
  *recvInplaceOffset = 0;
  *paramcount = *sendcount;
}

testResult_t BroadcastInitData(struct threadArgs* args, ncclDataType_t type, ncclRedOp_t op, int root, int rep, int in_place) {
  size_t sendcount = args->sendBytes / wordSize(type);
  size_t recvcount = args->expectedBytes / wordSize(type);

  for (int i=0; i<args->nGpus; i++) {
    CUDACHECK(hipSetDevice(args->gpus[i]));
    int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
    CUDACHECK(hipMemset(args->recvbuffs[i], 0, args->expectedBytes));
    void* data = in_place ? args->recvbuffs[i] : args->sendbuffs[i];
    if (rank == root) TESTCHECK(InitData(data, sendcount, 0, type, ncclSum, rep, 1, 0));
    TESTCHECK(InitData(args->expected[i], recvcount, 0, type, ncclSum, rep, 1, 0));
    CUDACHECK(hipDeviceSynchronize());
  }
  return testSuccess;
}

void BroadcastGetBw(size_t count, int typesize, double sec, double* algBw, double* busBw, int nranks) {
  double baseBw = (double)(count * typesize) / 1.0E9 / sec;

  *algBw = baseBw;
  double factor = 1;
  *busBw = baseBw * factor;
}

testResult_t BroadcastRunColl(void* sendbuff, void* recvbuff, size_t count, ncclDataType_t type, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream, struct threadArgs* args) {
  int rank;
  NCCLCHECK(ncclCommUserRank(comm, &rank));
#if NCCL_MAJOR >= 2 && NCCL_MINOR >= 2
  NCCLCHECK(ncclBroadcast(sendbuff, recvbuff, count, type, root, comm, stream));
#else
  if (rank == root) {
      NCCLCHECK(ncclBcast(sendbuff, count, type, root, comm, stream));
  } else {
      NCCLCHECK(ncclBcast(recvbuff, count, type, root, comm, stream));
  }
#endif
  return testSuccess;
}

struct testColl broadcastTest = {
  "Broadcast",
  BroadcastGetCollByteCount,
  BroadcastInitData,
  BroadcastGetBw,
  BroadcastRunColl
};

void BroadcastGetBuffSize(size_t *sendcount, size_t *recvcount, size_t count, int nranks) {
  size_t paramcount, sendInplaceOffset, recvInplaceOffset;
  BroadcastGetCollByteCount(sendcount, recvcount, &paramcount, &sendInplaceOffset, &recvInplaceOffset, count, nranks);
}

testResult_t BroadcastRunTest(struct threadArgs* args, int root, ncclDataType_t type, const char* typeName, ncclRedOp_t op, const char* opName) {
  args->collTest = &broadcastTest;
  ncclDataType_t *run_types;
  const char **run_typenames;
  int type_count;
  int begin_root, end_root;

  if ((int)type != -1) {
    type_count = 1;
    run_types = &type;
    run_typenames = &typeName;
  } else {
    type_count = test_typenum;
    run_types = test_types;
    run_typenames = test_typenames;
  }

  if (root != -1) {
    begin_root = end_root = root;
  } else {
    begin_root = 0;
    end_root = args->nProcs*args->nThreads*args->nGpus-1;
  }

  for (int i=0; i<type_count; i++) {
    for (int j=begin_root; j<=end_root; j++) {
      TESTCHECK(TimeTest(args, run_types[i], run_typenames[i], (ncclRedOp_t)0, "none", j));
    }
  }
  return testSuccess;
}

struct testEngine broadcastEngine = {
  BroadcastGetBuffSize,
  BroadcastRunTest
};

#pragma weak ncclTestEngine=broadcastEngine
