/*************************************************************************
 * Copyright (c) 2016-2022, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "hip/hip_runtime.h"
#include "common.h"

#define ALIGN 4

void ReduceScatterGetCollByteCount(size_t *sendcount, size_t *recvcount, size_t *paramcount, size_t *sendInplaceOffset, size_t *recvInplaceOffset, size_t count, int nranks) {
  size_t base = (count/(ALIGN*nranks))*ALIGN;
  *sendcount = base*nranks;
  *recvcount = base;
  *sendInplaceOffset = 0;
  *recvInplaceOffset = base;
  *paramcount = base;
}

testResult_t ReduceScatterInitData(struct threadArgs* args, ncclDataType_t type, ncclRedOp_t op, int root, int rep, int in_place) {
  size_t sendcount = args->sendBytes / wordSize(type);
  size_t recvcount = args->expectedBytes / wordSize(type);
  int nranks = args->nProcs*args->nThreads*args->nGpus;

  for (int i=0; i<args->nGpus; i++) {
    CUDACHECK(hipSetDevice(args->gpus[i]));
    int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
    CUDACHECK(hipMemset(args->recvbuffs[i], 0, args->expectedBytes));
    void* data = in_place ? args->recvbuffs[i] : args->sendbuffs[i];
    TESTCHECK(InitData(data, sendcount, 0, type, op, rep, nranks, rank));
    CUDACHECK(hipMemcpy(args->expected[i], args->recvbuffs[i], args->expectedBytes, hipMemcpyDefault));
    TESTCHECK(InitDataReduce(args->expected[i], recvcount, rank*recvcount, type, op, rep, nranks));
    CUDACHECK(hipDeviceSynchronize());
  }
  return testSuccess;
}

void ReduceScatterGetBw(size_t count, int typesize, double sec, double* algBw, double* busBw, int nranks) {
  double baseBw = (double)(count * typesize * nranks) / 1.0E9 / sec;

  *algBw = baseBw;
  double factor = ((double)(nranks - 1))/((double)nranks);
  *busBw = baseBw * factor;
}

testResult_t ReduceScatterRunColl(void* sendbuff, void* recvbuff, size_t count, ncclDataType_t type, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream, struct threadArgs* args) {
  NCCLCHECK(ncclReduceScatter(sendbuff, recvbuff, count, type, op, comm, stream));
  return testSuccess;
}

struct testColl reduceScatterTest = {
  "ReduceScatter",
  ReduceScatterGetCollByteCount,
  ReduceScatterInitData,
  ReduceScatterGetBw,
  ReduceScatterRunColl
};

void ReduceScatterGetBuffSize(size_t *sendcount, size_t *recvcount, size_t count, int nranks) {
  size_t paramcount, sendInplaceOffset, recvInplaceOffset;
  ReduceScatterGetCollByteCount(sendcount, recvcount, &paramcount, &sendInplaceOffset, &recvInplaceOffset, count, nranks);
}

testResult_t ReduceScatterRunTest(struct threadArgs* args, int root, ncclDataType_t type, const char* typeName, ncclRedOp_t op, const char* opName) {
  args->collTest = &reduceScatterTest;
  ncclDataType_t *run_types;
  ncclRedOp_t *run_ops;
  const char **run_typenames, **run_opnames;
  int type_count, op_count;

  if ((int)type != -1) {
    type_count = 1;
    run_types = &type;
    run_typenames = &typeName;
  } else {
    type_count = test_typenum;
    run_types = test_types;
    run_typenames = test_typenames;
  }

  if ((int)op != -1) {
    run_ops = &op;
    run_opnames = &opName;
    op_count = 1;
  } else {
    op_count = test_opnum;
    run_ops = test_ops;
    run_opnames = test_opnames;
  }

  for (int i=0; i<type_count; i++) {
    for (int j=0; j<op_count; j++) {
      TESTCHECK(TimeTest(args, run_types[i], run_typenames[i], run_ops[j], run_opnames[j], -1));
    }
  }
  return testSuccess;
}

struct testEngine reduceScatterEngine = {
  ReduceScatterGetBuffSize,
  ReduceScatterRunTest
};

#pragma weak ncclTestEngine=reduceScatterEngine
