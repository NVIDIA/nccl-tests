/*************************************************************************
 * Copyright (c) 2016-2022, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "hip/hip_runtime.h"
#include "common.h"

void BisectionGetCollByteCount(size_t *sendcount, size_t *recvcount, size_t *paramcount, size_t *sendInplaceOffset, size_t *recvInplaceOffset, size_t count, int nranks) {
  *sendcount = count;
  *recvcount = count;
  *sendInplaceOffset = 0;
  *recvInplaceOffset = 0;
  *paramcount = count;
}

int getPeer(int rank, int n_ranks){
    if (n_ranks % 4 == 0)
        return ((n_ranks / 2 + rank) % n_ranks) + (rank % 2 ? -1 : 1);
    // If there is an odd number of ranks, the last rank is ignored and paired with itself
    else if (n_ranks % 2 == 1 && rank == n_ranks-1)
        return rank;
    else
        return (rank + n_ranks/2) % (n_ranks - n_ranks % 2);
}

testResult_t BisectionInitData(struct threadArgs* args, ncclDataType_t type, ncclRedOp_t op, int root, int rep, int in_place) {
  size_t sendcount = args->sendBytes / wordSize(type);
  size_t recvcount = args->expectedBytes / wordSize(type);
  int nranks = args->nProcs*args->nThreads*args->nGpus;

  for (int i=0; i<args->nGpus; i++) {
    CUDACHECK(hipSetDevice(args->gpus[i]));
    int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
    CUDACHECK(hipMemset(args->recvbuffs[i], 0, args->expectedBytes));
    void* data = in_place ? args->recvbuffs[i] : args->sendbuffs[i];
    TESTCHECK(InitData(data, sendcount, rank*sendcount, type, ncclSum, rep, 1, 0));
    int peer = getPeer(rank, nranks);
    TESTCHECK(InitData(args->expected[i], recvcount, peer*recvcount, type, ncclSum, rep, 1, 0));
    CUDACHECK(hipDeviceSynchronize());
  }
  // We don't support in-place sendrecv
  args->reportErrors = in_place ? 0 : 1;
  return testSuccess;
}

void BisectionGetBw(size_t count, int typesize, double sec, double* algBw, double* busBw, int nranks) {
  *busBw = *algBw = (double)(count * typesize) / 1.0E9 / sec;
}

testResult_t BisectionRunColl(void* sendbuff, void* recvbuff, size_t count, ncclDataType_t type, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream) {
  int n_ranks, comm_rank, peer;

  NCCLCHECK(ncclCommUserRank(comm, &comm_rank));
  NCCLCHECK(ncclCommCount(comm, &n_ranks));

  peer = getPeer(comm_rank, n_ranks);

  NCCLCHECK(ncclGroupStart());
  NCCLCHECK(ncclSend(sendbuff, count, type, peer, comm, stream));
  NCCLCHECK(ncclRecv(recvbuff, count, type, peer, comm, stream));
  NCCLCHECK(ncclGroupEnd());

  return testSuccess;
}

struct testColl bisectionTest = {
  "Bisection",
  BisectionGetCollByteCount,
  BisectionInitData,
  BisectionGetBw,
  BisectionRunColl
};

void BisectionGetBuffSize(size_t *sendcount, size_t *recvcount, size_t count, int nranks) {
  size_t paramcount, sendInplaceOffset, recvInplaceOffset;
  BisectionGetCollByteCount(sendcount, recvcount, &paramcount, &sendInplaceOffset, &recvInplaceOffset, count, nranks);
}

testResult_t BisectionRunTest(struct threadArgs* args, int root, ncclDataType_t type, const char* typeName, ncclRedOp_t op, const char* opName) {
  args->collTest = &bisectionTest;
  ncclDataType_t *run_types;
  ncclRedOp_t *run_ops;
  const char **run_typenames, **run_opnames;
  int type_count, op_count;

  if ((int)type != -1) {
    type_count = 1;
    run_types = &type;
    run_typenames = &typeName;
  } else {
    type_count = test_typenum;
    run_types = test_types;
    run_typenames = test_typenames;
  }

  if ((int)op != -1) {
    op_count = 1;
    run_ops = &op;
    run_opnames = &opName;
  } else {
    op_count = test_opnum;
    run_ops = test_ops;
    run_opnames = test_opnames;
  }

  for (int i=0; i<type_count; i++) {
    for (int j=0; j<op_count; j++) {
      TESTCHECK(TimeTest(args, run_types[i], run_typenames[i], run_ops[j], run_opnames[j], -1));
    }
  }
  return testSuccess;
}

struct testEngine bisectionEngine = {
  BisectionGetBuffSize,
  BisectionRunTest
};

#pragma weak ncclTestEngine=bisectionEngine
