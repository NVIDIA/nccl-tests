/*************************************************************************
 * Copyright (c) 2016-2021, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "hip/hip_runtime.h"
#include "common.h"

void print_header() {
  PRINT("# %10s  %12s  %8s  %6s            out-of-place                       in-place          \n", "", "", "", "");
  PRINT("# %10s  %12s  %8s  %6s  %7s  %6s  %6s  %5s  %7s  %6s  %6s  %5s\n", "size", "count", "type", "root",
        "time", "algbw", "busbw", "error", "time", "algbw", "busbw", "error");
  PRINT("# %10s  %12s  %8s  %6s  %7s  %6s  %6s  %5s  %7s  %6s  %6s  %5s\n", "(B)", "(elements)", "", "",
        "(us)", "(GB/s)", "(GB/s)", "", "(us)", "(GB/s)", "(GB/s)", "");
}

void print_line_header (size_t size, size_t count, const char *typeName, const char *opName, int root) {
  PRINT("%12li  %12li  %8s  %6i", size, count, typeName, root);
}

void GatherGetCollByteCount(size_t *sendcount, size_t *recvcount, size_t *paramcount, size_t *sendInplaceOffset, size_t *recvInplaceOffset, size_t count, int nranks) {
  *sendcount = count/nranks;
  *recvcount = (count/nranks)*nranks;
  *sendInplaceOffset = count/nranks;
  *recvInplaceOffset = 0;
  *paramcount = count/nranks;
}

testResult_t GatherInitData(struct threadArgs* args, ncclDataType_t type, ncclRedOp_t op, int root, int rep, int in_place) {
  size_t sendcount = args->sendBytes / wordSize(type);
  size_t recvcount = args->expectedBytes / wordSize(type);
  int nranks = args->nProcs*args->nThreads*args->nGpus;
  int nGpusVisible;
  CUDACHECK(hipGetDeviceCount(&nGpusVisible));

  for (int i=0; i<args->nGpus; i++) {
    int gpuid = args->localRank*args->nThreads*args->nGpus + args->thread*args->nGpus + i;
    CUDACHECK(hipSetDevice(gpuid % nGpusVisible));
    int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
    CUDACHECK(hipMemset(args->recvbuffs[i], 0, args->expectedBytes));
    void* data = in_place ? ((char*)args->recvbuffs[i])+rank*args->sendBytes : args->sendbuffs[i];
    TESTCHECK(InitData(data, sendcount, type, rep, rank));
    CUDACHECK(hipMemcpy(args->expected[i], args->recvbuffs[i], args->expectedBytes, hipMemcpyDefault));
    if (rank == root) {
      for (int j=0; j<nranks; j++) {
        TESTCHECK(InitData(((char*)args->expected[i])+args->sendBytes*j, sendcount, type, rep, j));
      }
    }
    CUDACHECK(hipDeviceSynchronize());
  }
  return testSuccess;
}

void GatherGetBw(size_t count, int typesize, double sec, double* algBw, double* busBw, int nranks) {
  double baseBw = (double)(count * nranks * typesize) / 1.0E9 / sec;

  *algBw = baseBw;
  double factor = ((double)(nranks-1))/((double)(nranks));
  *busBw = baseBw * factor;
}

testResult_t GatherRunColl(void* sendbuff, void* recvbuff, size_t count, ncclDataType_t type, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream) {
  int nRanks;
  NCCLCHECK(ncclCommCount(comm, &nRanks));
  int rank;
  NCCLCHECK(ncclCommUserRank(comm, &rank));
  size_t rankOffset = count * wordSize(type);
  if (count == 0) return testSuccess;

  NCCLCHECK(ncclGroupStart());
  NCCLCHECK(ncclSend(sendbuff, count, type, root, comm, stream));
  if (rank == root) {
    for (int r=0; r<nRanks; r++) {
      NCCLCHECK(ncclRecv(((char*)recvbuff)+r*rankOffset, count, type, r, comm, stream));
    }
  }
  NCCLCHECK(ncclGroupEnd());

  return testSuccess;
}

struct testColl gatherTest = {
  "Gather",
  GatherGetCollByteCount,
  GatherInitData,
  GatherGetBw,
  GatherRunColl
};

void GatherGetBuffSize(size_t *sendcount, size_t *recvcount, size_t count, int nranks) {
  size_t paramcount, sendInplaceOffset, recvInplaceOffset;
  GatherGetCollByteCount(sendcount, recvcount, &paramcount, &sendInplaceOffset, &recvInplaceOffset, count, nranks);
}

testResult_t GatherRunTest(struct threadArgs* args, int root, ncclDataType_t type, const char* typeName, ncclRedOp_t op, const char* opName) {
  args->collTest = &gatherTest;
  ncclDataType_t *run_types;
  const char **run_typenames;
  int type_count;
  int begin_root, end_root;

  if ((int)type != -1) {
    type_count = 1;
    run_types = &type;
    run_typenames = &typeName;
  } else {
    type_count = test_typenum;
    run_types = test_types;
    run_typenames = test_typenames;
  }

  if (root != -1) {
    begin_root = end_root = root;
  } else {
    begin_root = 0;
    end_root = args->nProcs*args->nThreads*args->nGpus-1;
  }

  for (int i=0; i<type_count; i++) {
    for (int j=begin_root; j<=end_root; j++) {
      TESTCHECK(TimeTest(args, run_types[i], run_typenames[i], (ncclRedOp_t)0, "", j));
    }
  }
  return testSuccess;
}

struct testEngine gatherEngine = {
  GatherGetBuffSize,
  GatherRunTest
};

#pragma weak ncclTestEngine=gatherEngine
