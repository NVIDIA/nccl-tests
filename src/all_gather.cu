/*************************************************************************
 * Copyright (c) 2016-2022, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "hip/hip_runtime.h"
#include "common.h"

#define ALIGN 4

void AllGatherGetCollByteCount(size_t *sendcount, size_t *recvcount, size_t *paramcount, size_t *sendInplaceOffset, size_t *recvInplaceOffset, size_t count, int nranks) {
  size_t base = (count/(ALIGN*nranks))*ALIGN;
  *sendcount = base;
  *recvcount = base*nranks;
  *sendInplaceOffset = base;
  *recvInplaceOffset = 0;
  *paramcount = base;
}

testResult_t AllGatherInitData(struct threadArgs* args, ncclDataType_t type, ncclRedOp_t op, int root, int rep, int in_place) {
  size_t sendcount = args->sendBytes / wordSize(type);
  size_t recvcount = args->expectedBytes / wordSize(type);
  int nranks = args->nProcs*args->nThreads*args->nGpus;

  for (int i=0; i<args->nGpus; i++) {
    CUDACHECK(hipSetDevice(args->gpus[i]));
    int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
    CUDACHECK(hipMemset(args->recvbuffs[i], 0, args->expectedBytes));
    void* data = in_place ? ((char*)args->recvbuffs[i])+rank*args->sendBytes : args->sendbuffs[i];
    TESTCHECK(InitData(data, sendcount, 0, type, ncclSum, 33*rep + rank, 1, 0));
    for (int j=0; j<nranks; j++) {
      TESTCHECK(InitData((char*)args->expected[i] + args->sendBytes*j, sendcount, 0, type, ncclSum, 33*rep + j, 1, 0));
    }
    CUDACHECK(hipDeviceSynchronize());
  }
  return testSuccess;
}

void AllGatherGetBw(size_t count, int typesize, double sec, double* algBw, double* busBw, int nranks) {
  double baseBw = (double)(count * typesize * nranks) / 1.0E9 / sec;

  *algBw = baseBw;
  double factor = ((double)(nranks - 1))/((double)nranks);
  *busBw = baseBw * factor;
}

testResult_t AllGatherRunColl(void* sendbuff, void* recvbuff, size_t count, ncclDataType_t type, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream, struct threadArgs* args) {
  NCCLCHECK(ncclAllGather(sendbuff, recvbuff, count, type, comm, stream));
  return testSuccess;
}

struct testColl allGatherTest = {
  "AllGather",
  AllGatherGetCollByteCount,
  AllGatherInitData,
  AllGatherGetBw,
  AllGatherRunColl
};

void AllGatherGetBuffSize(size_t *sendcount, size_t *recvcount, size_t count, int nranks) {
  size_t paramcount, sendInplaceOffset, recvInplaceOffset;
  AllGatherGetCollByteCount(sendcount, recvcount, &paramcount, &sendInplaceOffset, &recvInplaceOffset, count, nranks);
}

testResult_t AllGatherRunTest(struct threadArgs* args, int root, ncclDataType_t type, const char* typeName, ncclRedOp_t op, const char* opName) {
  args->collTest = &allGatherTest;
  ncclDataType_t *run_types;
  const char **run_typenames;
  int type_count;

  if ((int)type != -1) {
    type_count = 1;
    run_types = &type;
    run_typenames = &typeName;
  } else {
    type_count = test_typenum;
    run_types = test_types;
    run_typenames = test_typenames;
  }

  for (int i=0; i<type_count; i++) {
    TESTCHECK(TimeTest(args, run_types[i], run_typenames[i], (ncclRedOp_t)0, "none", -1));
  }
  return testSuccess;
}

struct testEngine allGatherEngine = {
  AllGatherGetBuffSize,
  AllGatherRunTest
};

#pragma weak ncclTestEngine=allGatherEngine
