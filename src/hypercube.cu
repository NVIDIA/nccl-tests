/*************************************************************************
 * Copyright (c) 2016-2022, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "hip/hip_runtime.h"
#include "common.h"

#define ALIGN 4

void HyperCubeGetCollByteCount(size_t *sendcount, size_t *recvcount, size_t *paramcount, size_t *sendInplaceOffset, size_t *recvInplaceOffset, size_t count, int nranks) {
  size_t base = (count/(ALIGN*nranks))*ALIGN;
  *sendcount = base;
  *recvcount = base*nranks;
  *sendInplaceOffset = base;
  *recvInplaceOffset = 0;
  *paramcount = base;
}

testResult_t HyperCubeInitData(struct threadArgs* args, ncclDataType_t type, ncclRedOp_t op, int root, int rep, int in_place) {
  size_t sendcount = args->sendBytes / wordSize(type);
  size_t recvcount = args->expectedBytes / wordSize(type);
  int nranks = args->nProcs*args->nThreads*args->nGpus;

  for (int i=0; i<args->nGpus; i++) {
    CUDACHECK(hipSetDevice(args->gpus[i]));
    int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
    CUDACHECK(hipMemset(args->recvbuffs[i], 0, args->expectedBytes));
    void* data = in_place ? ((char*)args->recvbuffs[i])+rank*args->sendBytes : args->sendbuffs[i];
    TESTCHECK(InitData(data, sendcount, 0, type, ncclSum, 33*rep + rank, 1, 0));
    for (int j=0; j<nranks; j++) {
      TESTCHECK(InitData((char*)args->expected[i] + args->sendBytes*j, sendcount, 0, type, ncclSum, 33*rep + j, 1, 0));
    }
    CUDACHECK(hipDeviceSynchronize());
  }
  return testSuccess;
}

void HyperCubeGetBw(size_t count, int typesize, double sec, double* algBw, double* busBw, int nranks) {
  double baseBw = (double)(count * typesize * (nranks - 1)) / 1.0E9 / sec;

  *algBw = baseBw;
  double factor = 1;
  *busBw = baseBw * factor;
}

testResult_t HyperCubeRunColl(void* sendbuff, void* recvbuff, size_t count, ncclDataType_t type, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream, struct threadArgs* args) {
  char* sbuff = (char*)sendbuff;
  char* rbuff = (char*)recvbuff;
  int nRanks;
  NCCLCHECK(ncclCommCount(comm, &nRanks));
  int rank;
  NCCLCHECK(ncclCommUserRank(comm, &rank));
  size_t rankSize = count * wordSize(type);

  if (rbuff+rank*rankSize != sbuff) CUDACHECK(hipMemcpyAsync(rbuff+rank*rankSize, sbuff, rankSize, hipMemcpyDeviceToDevice, stream));

  // Hypercube AllGather
  for (int mask=1; mask<nRanks; mask<<=1) {
    NCCLCHECK(ncclGroupStart());
    int s = rank & ~(mask-1);
    int r = s ^ mask;
    NCCLCHECK(ncclSend(rbuff+s*rankSize, count*mask, type, rank^mask, comm, stream));
    NCCLCHECK(ncclRecv(rbuff+r*rankSize, count*mask, type, rank^mask, comm, stream));
    NCCLCHECK(ncclGroupEnd());
  }
  return testSuccess;
}

struct testColl hyperCubeTest = {
  "HyperCube",
  HyperCubeGetCollByteCount,
  HyperCubeInitData,
  HyperCubeGetBw,
  HyperCubeRunColl
};

void HyperCubeGetBuffSize(size_t *sendcount, size_t *recvcount, size_t count, int nranks) {
  size_t paramcount, sendInplaceOffset, recvInplaceOffset;
  HyperCubeGetCollByteCount(sendcount, recvcount, &paramcount, &sendInplaceOffset, &recvInplaceOffset, count, nranks);
}

testResult_t HyperCubeRunTest(struct threadArgs* args, int root, ncclDataType_t type, const char* typeName, ncclRedOp_t op, const char* opName) {
  args->collTest = &hyperCubeTest;
  ncclDataType_t *run_types;
  const char **run_typenames;
  int type_count;

  if ((int)type != -1) {
    type_count = 1;
    run_types = &type;
    run_typenames = &typeName;
  } else {
    type_count = test_typenum;
    run_types = test_types;
    run_typenames = test_typenames;
  }

  // Check if this is a power of 2
  int nRanks = args->nProcs*args->nThreads*args->nGpus;
  if (nRanks && !(nRanks & (nRanks - 1))) {
    for (int i=0; i<type_count; i++) {
      TESTCHECK(TimeTest(args, run_types[i], run_typenames[i], (ncclRedOp_t)0, "", -1));
    }
  } else {
    printf("nRanks %d is not a power of 2, skipping\n", nRanks);
  }

  return testSuccess;
}

struct testEngine hyperCubeEngine = {
  HyperCubeGetBuffSize,
  HyperCubeRunTest
};

#pragma weak ncclTestEngine=hyperCubeEngine
