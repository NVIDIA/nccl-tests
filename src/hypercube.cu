/*************************************************************************
 * Copyright (c) 2016-2021, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "hip/hip_runtime.h"
#include "common.h"

#define ALIGN 4

void print_header() {
  PRINT("# %10s  %12s  %8s            out-of-place                       in-place          \n", "", "", "");
  PRINT("# %10s  %12s  %8s  %7s  %6s  %6s  %5s  %7s  %6s  %6s  %5s\n", "size", "count", "type",
        "time", "algbw", "busbw", "error", "time", "algbw", "busbw", "error");
  PRINT("# %10s  %12s  %8s  %7s  %6s  %6s  %5s  %7s  %6s  %6s  %5s\n", "(B)", "(elements)", "",
        "(us)", "(GB/s)", "(GB/s)", "", "(us)", "(GB/s)", "(GB/s)", "");
}

void print_line_header (size_t size, size_t count, const char *typeName, const char *opName, int root) {
  PRINT("%12li  %12li  %8s", size, count, typeName);
}

void HyperCubeGetCollByteCount(size_t *sendcount, size_t *recvcount, size_t *paramcount, size_t *sendInplaceOffset, size_t *recvInplaceOffset, size_t count, int nranks) {
  size_t base = (count/(ALIGN*nranks))*ALIGN;
  *sendcount = base;
  *recvcount = base*nranks;
  *sendInplaceOffset = base;
  *recvInplaceOffset = 0;
  *paramcount = base;
}

testResult_t HyperCubeInitData(struct threadArgs* args, ncclDataType_t type, ncclRedOp_t op, int root, int rep, int in_place) {
  size_t sendcount = args->sendBytes / wordSize(type);
  size_t recvcount = args->expectedBytes / wordSize(type);
  int nranks = args->nProcs*args->nThreads*args->nGpus;
  int nGpusVisible;
  CUDACHECK(hipGetDeviceCount(&nGpusVisible));

  for (int i=0; i<args->nGpus; i++) {
    int gpuid = args->localRank*args->nThreads*args->nGpus + args->thread*args->nGpus + i;
    CUDACHECK(hipSetDevice(gpuid % nGpusVisible));
    int rank = ((args->proc*args->nThreads + args->thread)*args->nGpus + i);
    CUDACHECK(hipMemset(args->recvbuffs[i], 0, args->expectedBytes));
    void* data = in_place ? ((char*)args->recvbuffs[i])+rank*args->sendBytes : args->sendbuffs[i];
    TESTCHECK(InitData(data, sendcount, type, rep, rank));
    for (int j=0; j<nranks; j++) {
      TESTCHECK(InitData(((char*)args->expected[i])+args->sendBytes*j, sendcount, type, rep, j));
    }
    CUDACHECK(hipDeviceSynchronize());
  }
  return testSuccess;
}

void HyperCubeGetBw(size_t count, int typesize, double sec, double* algBw, double* busBw, int nranks) {
  double baseBw = (double)(count * typesize * (nranks - 1)) / 1.0E9 / sec;

  *algBw = baseBw;
  double factor = 1;
  *busBw = baseBw * factor;
}

testResult_t HyperCubeRunColl(void* sendbuff, void* recvbuff, size_t count, ncclDataType_t type, ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream) {
  char* sbuff = (char*)sendbuff;
  char* rbuff = (char*)recvbuff;
  int nRanks;
  NCCLCHECK(ncclCommCount(comm, &nRanks));
  int rank;
  NCCLCHECK(ncclCommUserRank(comm, &rank));
  size_t rankSize = count * wordSize(type);

  if (rbuff+rank*rankSize != sbuff) CUDACHECK(hipMemcpyAsync(rbuff+rank*rankSize, sbuff, rankSize, hipMemcpyDeviceToDevice, stream));

  // Hypercube AllGather
  for (int mask=1; mask<nRanks; mask<<=1) {
    NCCLCHECK(ncclGroupStart());
    int s = rank & ~(mask-1);
    int r = s ^ mask;
    NCCLCHECK(ncclSend(rbuff+s*rankSize, count*mask, type, rank^mask, comm, stream));
    NCCLCHECK(ncclRecv(rbuff+r*rankSize, count*mask, type, rank^mask, comm, stream));
    NCCLCHECK(ncclGroupEnd());
  }
  return testSuccess;
}

struct testColl hyperCubeTest = {
  "HyperCube",
  HyperCubeGetCollByteCount,
  HyperCubeInitData,
  HyperCubeGetBw,
  HyperCubeRunColl
};

void HyperCubeGetBuffSize(size_t *sendcount, size_t *recvcount, size_t count, int nranks) {
  size_t paramcount, sendInplaceOffset, recvInplaceOffset;
  HyperCubeGetCollByteCount(sendcount, recvcount, &paramcount, &sendInplaceOffset, &recvInplaceOffset, count, nranks);
}

testResult_t HyperCubeRunTest(struct threadArgs* args, int root, ncclDataType_t type, const char* typeName, ncclRedOp_t op, const char* opName) {
  args->collTest = &hyperCubeTest;
  ncclDataType_t *run_types;
  const char **run_typenames;
  int type_count;

  if ((int)type != -1) {
    type_count = 1;
    run_types = &type;
    run_typenames = &typeName;
  } else {
    type_count = test_typenum;
    run_types = test_types;
    run_typenames = test_typenames;
  }

  for (int i=0; i<type_count; i++) {
    TESTCHECK(TimeTest(args, run_types[i], run_typenames[i], (ncclRedOp_t)0, "", -1));
  }
  return testSuccess;
}

struct testEngine hyperCubeEngine = {
  HyperCubeGetBuffSize,
  HyperCubeRunTest
};

#pragma weak ncclTestEngine=hyperCubeEngine
